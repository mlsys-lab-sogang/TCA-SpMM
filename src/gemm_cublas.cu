#include "gemm_cublas.cuh"

void perform_GemmEx(half *A, half *B, float *C, int M, int N, int K, int n_iter, double *elapsed = NULL)
{
    warm_up_gpu_kernel<<<1, 32>>>();
    hipDeviceSynchronize();

    hipblasHandle_t handle;

    float measured_time = 0.0;
    float alloc_time, h2d_time, d2h_time;
    alloc_time = h2d_time = d2h_time = 0.0;
    half *dA, *dB;
    float *dC;
    size_t total_mem = 0;

    float alpha = 1.0f;
    float beta = 0.0f;
    hipDataType dtypeA, dtypeB, dtypeC;
    dtypeA = dtypeB = HIP_R_16F;
    dtypeC = HIP_R_32F;
    hipblasComputeType_t comptype;
    comptype = HIPBLAS_COMPUTE_32F;
    hipEvent_t start, end;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&end));
    VERBOSE_PUTS("Preparing Resources...");
    CHECK_CUDA(hipEventRecord(start));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(A[0]) * M * K));
    total_mem += sizeof(A[0]) * M * K;
    CHECK_CUDA(hipMalloc((void **)&dB, sizeof(B[0]) * K * N));
    total_mem += sizeof(B[0]) * K * N;
    CHECK_CUDA(hipMalloc((void **)&dC, sizeof(C[0]) * M * N));
    total_mem += sizeof(C[0]) * M * N;

    CHECK_CUBLAS(hipblasCreate(&handle));
    // CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    CHECK_CUDA(hipEventRecord(end));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventElapsedTime(&alloc_time, start, end));
    VERBOSE_PRINT("Done %fms taken\n", alloc_time);

    VERBOSE_PUTS("H2D Memcpy...");
    CHECK_CUDA(hipEventRecord(start));
    CHECK_CUBLAS(hipblasSetVector(M * K, sizeof(dA[0]), A, 1, dA, 1));
    CHECK_CUBLAS(hipblasSetVector(K * N, sizeof(dB[0]), B, 1, dB, 1));
    CHECK_CUBLAS(hipblasSetVector(M * N, sizeof(dC[0]), C, 1, dC, 1));

    // CHECK_CUDA(hipMemcpy(dA, A, sizeof(A[0]) * M * K, hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy(dC, B, sizeof(B[0]) * K * N, hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy(dC, C, sizeof(C[0]) * M * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipEventRecord(end));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventElapsedTime(&h2d_time, start, end));

    VERBOSE_PRINT("Done %fms taken\n", h2d_time);

    for (int i = 0; i < 5; i++)
    {
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  N, M, K,
                                  &alpha,
                                  dB, dtypeB, N,
                                  dA, dtypeA, K,
                                  &beta,
                                  dC, dtypeC, N,
                                  comptype, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }

    CHECK_CUDA(hipDeviceSynchronize());
    VERBOSE_PUTS("Performs GemmEx...");
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < n_iter; i++)
    {
        // obtain C^T, instead of C, to utilize col-major system.
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  N, M, K,
                                  &alpha,
                                  dB, dtypeB, N,
                                  dA, dtypeA, K,
                                  &beta,
                                  dC, dtypeC, N,
                                  comptype, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    CHECK_CUDA(hipEventRecord(end));
    CHECK_CUDA(hipEventSynchronize(end));

    CHECK_CUDA(hipEventElapsedTime(&measured_time, start, end));
    VERBOSE_PRINT("Done %fms taken\n", measured_time);
    measured_time = measured_time / (float)n_iter;

    VERBOSE_PUTS("D2H  Memcpy...");
    CHECK_CUDA(hipEventRecord(start));
    CHECK_CUBLAS(hipblasGetVector(M * N, sizeof(C[0]), dC, 1, C, 1));
    CHECK_CUDA(hipEventRecord(end));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventElapsedTime(&d2h_time, start, end));
    VERBOSE_PRINT("Done %fms taken\n", d2h_time);

    if (elapsed != NULL)
    {
        *elapsed = (double)measured_time;
    }

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(end));

    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
}